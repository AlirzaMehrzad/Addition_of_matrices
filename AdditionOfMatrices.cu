#include "hip/hip_runtime.h"

 #include <stdio.h>
 #include <stdlib.h>
 #include <math.h>
 
 /*---------------------------------------------------------------------
  * Kernel:   AdditionOfMatrices
  * Purpose:  Implement matrix addition
  * In args:  A, B, m, n
  * Out arg:  C
  */
 __global__ void AdditionOfMatrices(float A[], float B[], float C[], int m, int n) {
    /* blockDim.x = threads_per_block                            */
    /* First block gets first threads_per_block components.      */
    /* Second block gets next threads_per_block components, etc. */
    int my_ij = blockDim.x * blockIdx.x + threadIdx.x;
 
    /* The test shouldn't be necessary */
    if (blockIdx.x < m && threadIdx.x < n) 
       C[my_ij] = A[my_ij] + B[my_ij];
 }  /* AdditionOfMatrices */
 
 
 /*---------------------------------------------------------------------
  * Function:  Read_matrix
  * Purpose:   Read an m x n matrix from stdin
  * In args:   m, n
  * Out arg:   A
  */
  void Read_matrix(float A[], int m, int n) {
   int i, j;

   for (i = 0; i < m; i++)
      for (j = 0; j < n; j++)
         scanf("%f", &A[i*n+j]);
}  /* Read_matrix */


/*---------------------------------------------------------------------
 * Function:  Print_matrix
 * Purpose:   Print an m x n matrix to stdout
 * In args:   title, A, m, n
 */
void Print_matrix(char title[], float A[], int m, int n) {
   int i, j;

   printf("%s\n", title);
   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.1f ", A[i*n+j]);
      printf("\n");
   }  
}  /* Print_matrix */
 
 
 /* Host code */
 int main(int argc, char* argv[]) {
    int m, n;
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    size_t size;
 
    /* Get size of matrices */
    if (argc != 3) {
       fprintf(stderr, "usage: %s <row count> <col count>\n", argv[0]);
       exit(0);
    }
    m = strtol(argv[1], NULL, 10);
    n = strtol(argv[2], NULL, 10);
    printf("m = %d, n = %d\n", m, n);
    size = m*n*sizeof(float);
 
    h_A = (float*) malloc(size);
    h_B = (float*) malloc(size);
    h_C = (float*) malloc(size);
    
    printf("Enter the matrices A and B\n");
    Read_matrix(h_A, m, n);
    printf("matrix a is complete, now enter secend matrix: ");
    Read_matrix(h_B, m, n);
 
    Print_matrix("A =", h_A, m, n);
    Print_matrix("B =", h_B, m, n);
 
    /* Allocate matrices in device memory */
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
 
    /* Copy matrices from host memory to device memory */
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
 
    /* Invoke kernel using m thread blocks, each of    */
    /* which contains n threads */
    /* Execution configuration */
    dim3 block_size( 16, 16 );
    dim3 num_blocks( ( n - 1 + block_size.x ) / block_size.x, 
                     ( m - 1 + block_size.y ) / block_size.y );
  
    AdditionOfMatrices<<<block_size, num_blocks>>>(d_A, d_B, d_C, m, n);
 
    /* Wait for the kernel to complete */
    hipDeviceSynchronize();
 
    /* Copy result from device memory to host memory */
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
 
    Print_matrix("The sum is: ", h_C, m, n);
 
    /* Free device memory */
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
 
    /* Free host memory */
    free(h_A);
    free(h_B);
    free(h_C);
 
    return 0;
 }  /* main */